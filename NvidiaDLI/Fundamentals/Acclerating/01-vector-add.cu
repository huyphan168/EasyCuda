
#include <hip/hip_runtime.h>
#include <stdio.h>

inline hipError_t checkCuda(hipError_t result){
    if (result != hipSuccess){
        fprintf(stderr, "Error: %s\n", hipGetErrorString(result));
    }
    return result;
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i+=stride){
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
    const int N = 2<<20;
    size_t size = N * sizeof(float);

    float *a;
    float *b;
    float *c;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    initWith(3, a, N);
    initWith(4, b, N);
    initWith(0, c, N);
    
    int threads_per_block = 1024;
    int number_of_blocks = N < 1024*32 ? (N+threads_per_block-1)/threads_per_block : 32;

    addVectorsInto<<<number_of_blocks, threads_per_block>>>(c, a, b, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        printf("Error: %s", hipGetErrorString(err));
    }
    else printf("Success!");
    checkCuda(hipDeviceSynchronize());
    checkElementsAre(7, c, N);
    hipFree(a);
    hipFree(b);
    hipFree(c);
}
