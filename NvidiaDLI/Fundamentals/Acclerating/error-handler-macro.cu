
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<assert.h>

inline hipError_t checkCuda(hipError_t result){
    if (result != hipSuccess){
        fprintf(stderr, "Error: %s\n", hipGetErrorString(result));
    }
    return result;
}

int main(void){
    checkCuda(hipDeviceSynchronize());
}