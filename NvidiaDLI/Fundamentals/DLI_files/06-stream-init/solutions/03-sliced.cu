
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;
  const int numberOfStreams = 8;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  const int streamN = N / numberOfStreams;

  size_t streamSize = streamN * sizeof(float);

  float *a[numberOfStreams];
  float *b[numberOfStreams];
  float *c[numberOfStreams];
  float *h_c[numberOfStreams];

  hipStream_t streams[numberOfStreams];
  hipStream_t memoryStream;

  hipStreamCreate(&memoryStream);

  for (int i = 0; i < numberOfStreams; ++i)
  {
    hipStreamCreate(&streams[i]);
    hipMalloc(&a[i], streamSize);
    hipMalloc(&b[i], streamSize);
    hipMalloc(&c[i], streamSize);
    h_c[i] = (float *)malloc(streamSize);
  }

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  for (int i = 0; i < numberOfStreams; ++i)
  {
    initWith<<<numberOfBlocks, threadsPerBlock, 0, streams[i]>>>(3, a[i], streamN);
    initWith<<<numberOfBlocks, threadsPerBlock, 0, streams[i]>>>(4, b[i], streamN);
    initWith<<<numberOfBlocks, threadsPerBlock, 0, streams[i]>>>(0, c[i], streamN);

    addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, streams[i]>>>(c[i], a[i], b[i], streamN);

    hipMemcpyAsync(h_c[i], c[i], streamSize, hipMemcpyDeviceToHost, streams[i]);
    // cudaMemcpyAsync(h_c[i], c[i], streamSize, cudaMemcpyDeviceToHost, memoryStream);
  }

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));


  /*
   * Destroy streams when they are no longer needed.
   */

  for (int i = 0; i < numberOfStreams; ++i)
  {
    checkElementsAre(7, h_c[i], streamN);
    hipStreamDestroy(streams[i]);
    hipFree(a[i]);
    hipFree(b[i]);
    hipFree(c[i]);
    free(h_c[i]);
  }
  hipStreamDestroy(memoryStream);

}
