
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
  float *h_c;

  hipMalloc(&a, size);
  hipMalloc(&b, size);
  hipMalloc(&c, size);
  hipHostMalloc(&h_c, size, hipHostMallocDefault);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErrInit;
  hipError_t asyncErrAdd;

  /*
   * Create 3 streams to run initialize the 3 data vectors in parallel.
   */

  hipStream_t stream1, stream2, stream3;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  /*
   * Give each `initWith` launch its own non-standard stream.
   */

  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, N);

  /*
   * Synchronize - we are using different non-standard streams for initialisation
   * and calcs, so we need to explicitly synchronize to ensure all data has been
   * initialised before starting the calculations 
   */
  asyncErrInit = hipDeviceSynchronize();
  if(asyncErrInit != hipSuccess) printf("Error init: %s\n", hipGetErrorString(asyncErrInit));

  for (int i = 0; i < 4; ++i)
  {
    hipStream_t stream;
    hipStreamCreate(&stream);

    addVectorsInto<<<numberOfBlocks/4, threadsPerBlock, 0, stream>>>(&c[i*N/4], &a[i*N/4], &b[i*N/4], N/4);
    hipMemcpyAsync(&h_c[i*N/4], &c[i*N/4], size/4, hipMemcpyDeviceToHost, stream);
    hipStreamDestroy(stream);
  }

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErrAdd = hipDeviceSynchronize();
  if(asyncErrAdd != hipSuccess) printf("Error add: %s\n", hipGetErrorString(asyncErrAdd));

  checkElementsAre(7, h_c, N);

  /*
   * Destroy streams when they are no longer needed.
   */

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);

  hipFree(a);
  hipFree(b);
  hipFree(c);
  hipHostFree(h_c);
}
