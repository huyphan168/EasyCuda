#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
    int tid = blockIdx.x * blockDim.x * threadIdx.x;

    if ( tid < N )
        c[tid] = 2 * a[tid] + b[tid];
}

int main()
{
    float *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // Initialize memory
    for( int i = 0; i < N; ++i )
    {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }

    int threads_per_block = 128;
    int number_of_blocks = (N / threads_per_block) + 1;

    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
