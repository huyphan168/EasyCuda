#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */
__global__
void initWith(float num, float *a, int N)
{
  
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < N) { a[index] = num; }

}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  /*
   * nsys should register performance changes when execution configuration
   * is updated.
   */

  threadsPerBlock = 1024;
  numberOfBlocks = 40;

  int deviceId;
  hipGetDevice(&deviceId);
  
  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);

  initWith<<<numberOfBlocks*N/threadsPerBlock+1, threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks*N/threadsPerBlock+1, threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks*N/threadsPerBlock+1, threadsPerBlock>>>(0, c, N);

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
  hipDeviceSynchronize();

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
  hipMemPrefetchAsync(c, size, hipCpuDeviceId);
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}